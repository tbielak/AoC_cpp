#include "hip/hip_runtime.h"
// puzzle input is kept in CUDA device constant memory
// maximum input length is defined here
__constant__ int day_24_2015_CUDA_input[32];

// CUDA device kernel code
__global__ void Day_24_2015_CUDA_findGroups_kernel(int groupsPerThread, unsigned char* results, int packages_count, int packages_weight)
{
    // thread index
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // starting group
    int starting_group = idx * groupsPerThread;
    
    // check groups
    for (int g = 0; g < groupsPerThread; g++)
    {
        // check single group
        int group = starting_group + g;

        // find weight of the group and count number of packages in the group (bits set)
        int weight = 0;
        unsigned char number_of_packages = 0;
        for (int i = packages_count - 1; i >= 0; i--)
        {
            if (group & 1)
            {
                number_of_packages++;
                weight += day_24_2015_CUDA_input[i];
                if (weight > packages_weight)
                    break;
            }

            group >>= 1;
        }

        if (weight != packages_weight)
            number_of_packages = 0xff;      // perfect weight not found

        // perfect weight found -> store number of packages of the group
        results[starting_group + g] = number_of_packages;
    }
}

// CUDA device kernel code
__global__ void Day_24_2015_CUDA_findGroupsMin_kernel(int groupsPerThread, unsigned char* results, unsigned char* aggregated)
{
    // thread index
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // starting group
    int starting_group = idx * groupsPerThread;

    // find minimum number of packages in the group
    unsigned char number_of_packages = 0xff;
    for (int g = 0; g < groupsPerThread; g++)
        if (number_of_packages > results[starting_group + g])
            number_of_packages = results[starting_group + g];

    aggregated[idx] = number_of_packages;
}

// CUDA device kernel code
__global__ void Day_24_2015_CUDA_findQEMin_kernel(int groupsPerThread, unsigned char* results, long long* minQE_results, int packages_count, unsigned char minimum_packages_count)
{
    // result
    long long minQE = 0x7fffffffffffffff;

    // thread index
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // starting group
    int starting_group = idx * groupsPerThread;

    // check groups
    for (int g = 0; g < groupsPerThread; g++)
    {
        // check single group
        int group = starting_group + g;

        // check if group has requested number of packages
        if (results[group] != minimum_packages_count)
            continue;

        // calculate QE
        long long v = 1;
        for (int i = packages_count - 1; i >= 0; i--)
        {
            if (group & 1)
                v *= day_24_2015_CUDA_input[i];

            group >>= 1;
        }

        // is it minQE so far?
        if (minQE > v) 
            minQE = v;
    }

    // return minQE of groupsPerThread groups
    minQE_results[idx] = minQE;
}

// host code: storing input in device constant memory
bool Day_04_2015_CUDA_input(const int input[32])
{
    return hipMemcpyToSymbol(HIP_SYMBOL(day_24_2015_CUDA_input), input, 4 * 32) == hipSuccess;
}

// host code: calling CUDA device kernel code
bool Day_24_2015_CUDA_findGroups(int blocksPerGrid, int threadsPerBlock, int groupsPerThread, unsigned char* results, int packages_count, int packages_weight)
{
    Day_24_2015_CUDA_findGroups_kernel << <blocksPerGrid, threadsPerBlock >> > (groupsPerThread, results, packages_count, packages_weight);
    bool status = (hipGetLastError() == hipSuccess);
    
    hipDeviceSynchronize();
    return status;
}

// host code: calling CUDA device kernel code
bool Day_24_2015_CUDA_findGroupsMin(int blocksPerGrid, int threadsPerBlock, int groupsPerThread, unsigned char* results, unsigned char* aggregated)
{
    Day_24_2015_CUDA_findGroupsMin_kernel << <blocksPerGrid, threadsPerBlock >> > (groupsPerThread, results, aggregated);
    bool status = (hipGetLastError() == hipSuccess);

    hipDeviceSynchronize();
    return status;
}

// host code: calling CUDA device kernel code
bool Day_24_2015_CUDA_findQEMin(int blocksPerGrid, int threadsPerBlock, int groupsPerThread, unsigned char* results, long long* minQE_results, int packages_count, unsigned char minimum_packages_count)
{
    Day_24_2015_CUDA_findQEMin_kernel << <blocksPerGrid, threadsPerBlock >> > (groupsPerThread, results, minQE_results, packages_count, minimum_packages_count);
    bool status = (hipGetLastError() == hipSuccess);

    hipDeviceSynchronize();
    return status;
}
