#include "hip/hip_runtime.h"
// puzzle input is kept in CUDA device constant memory
// maximum input length is defined here
__constant__ char day_04_2015_CUDA_input[16];

// real input length
__constant__ int day_04_2015_CUDA_input_length;

// below: including single-MD5-transform routine shared by two CPU solutions and one GPU solution
// note: GPU version requires prefix __device__ placing the code on GPU side
#define PREFIX_2015_04 __device__
#define FNAME_2015_04 Day_04_2015_CUDA_md5_single_transform
#include "2015_04_shared.h"

// CUDA device kernel code
__global__ void Day_04_2015_CUDA_md5Check_kernel(unsigned char* results, unsigned int result_mask, int starting_point)
{
	// prepare 64-byte buffer with input
    char buffer[64];

    // copy puzzle input
    int i = 0;
    for (; i < day_04_2015_CUDA_input_length; i++)
        buffer[i] = day_04_2015_CUDA_input[i];
    
    // add appropriate number basing on thread index
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int number = starting_point + idx;

    int j = i;
    while (number > 0)
    {
        for (int k = i; k > j; k--)
            buffer[k] = buffer[k - 1];

        buffer[j] = (number % 10) + '0';
        number /= 10;
        i++;
    }

    // MD5 padding with length
    int length = i * 8;
    if (i < 64)
        buffer[i++] = -128;

    for (; i < 64; i++)
        buffer[i] = 0;

    *((int*)(&buffer[56])) = length;

    // do single MD5 transform and take part of the digest
    unsigned int result = FNAME_2015_04((unsigned int*)buffer);

    // check requested number of zeros and store result
    results[idx] = ((result & result_mask) == result) ? 1 : 0;
}

// host code: storing input in device constant memory
bool Day_04_2015_CUDA_input(const char* input, int input_length)
{
    if (hipMemcpyToSymbol(HIP_SYMBOL(day_04_2015_CUDA_input), input, input_length) != hipSuccess)
        return false;
    
    if (hipMemcpyToSymbol(HIP_SYMBOL(day_04_2015_CUDA_input_length), &input_length, 4) != hipSuccess)
        return false;

    return true;
}

// host code: calling CUDA device kernel code
bool Day_04_2015_CUDA_md5Check(int blocksPerGrid, int threadsPerBlock, unsigned char* results, unsigned int result_mask, int starting_point)
{
	Day_04_2015_CUDA_md5Check_kernel << <blocksPerGrid, threadsPerBlock >> > (results, result_mask, starting_point);
	return (hipGetLastError() == hipSuccess);
}

// undefine macros (different definitions for CPU/GPU version)
#undef PREFIX_2015_04
#undef FNAME_2015_04
